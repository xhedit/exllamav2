#include "hip/hip_runtime.h"
#include "rms_norm.cuh"
#include "util.cuh"
#include "compat.cuh"

#if defined(USE_ROCM)
#define NUM_WARPS (1024 / warpSize)
#define WARP_SIZE (warpSize)
#else
#define NUM_WARPS 32
#define WARP_SIZE 32
#endif

// y = x * w / sqrt(row_mean(x * x) + epsilon)

#define BLOCK_SIZE WARP_SIZE
#define NUM_THREADS (NUM_WARPS * WARP_SIZE)

typedef void (*fp_rms_norm_kernel)
(
    const half*,
    const half*,
    half*,
    const float,
    const float,
    const int,
    const int
);

template <int blocks_per_warp>
__global__ void rms_norm_kernel
(
    const half* __restrict__ x,
    const half* __restrict__ w,
    half* __restrict__ y,
    const float epsilon,
    const float r_dim,
    const int rows,
    const int dim
)
{
    int warp_id = threadIdx.x / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int row = blockIdx.x;
    const half2* x_row = (const half2*) (x + row * dim);
    half2* y_row = (half2*) (y + row * dim);
    const half2* w2 = (const half2*) w;

    // Compute sum of squares for each block

    float sum = 0.0f;
    float itemf[blocks_per_warp][2];

    #pragma unroll
    for (int i = 0; i < blocks_per_warp; i++)
    {
        int column = warp_id * WARP_SIZE + lane_id + NUM_THREADS * i;
        if (column >= dim / 2) break;

        half2 x2 = x_row[column];
        float f0 = __half2float(__low2half(x2));
        float f1 = __half2float(__high2half(x2));
        f0 = fmaxf(-65504.0f, fminf(f0, 65504.0f));
        f1 = fmaxf(-65504.0f, fminf(f1, 65504.0f));
        itemf[i][0] = f0;
        itemf[i][1] = f1;
        sum = fma(f0, f0, sum);
        sum = fma(f1, f1, sum);
    }

    // Shuffle to sum across lanes

    __shared__ float sums[NUM_WARPS];

    for(int offset = warpSize / 2; offset > 0; offset /= 2) sum += __shfl_xor_sync(0xffffffff, sum, offset);
    if (lane_id == 0) sums[warp_id] = sum;
    __syncthreads();

    // Load partial sums from across warps, shuffle again across lanes

    sum = sums[lane_id];
    for(int offset = warpSize / 2; offset > 0; offset /= 2) sum += __shfl_xor_sync(0xffffffff, sum, offset);

    // Get norm

    float rmf = rsqrtf(sum * r_dim + epsilon);

    // Normalize x, scaling by w

    #pragma unroll
    for (int i = 0; i < blocks_per_warp; i++)
    {
        int column = warp_id * WARP_SIZE + lane_id + NUM_THREADS * i;
        if (column >= dim / 2) return;
        half2 w2_ = w2[column];

        float x_itemf0 = itemf[i][0];
        float x_itemf1 = itemf[i][1];
        float w_itemf0 = __half2float(__low2half(w2_));
        float w_itemf1 = __half2float(__high2half(w2_));
        float n0 = x_itemf0 * w_itemf0 * rmf;
        float n1 = x_itemf1 * w_itemf1 * rmf;
        y_row[column] = __halves2half2(__float2half_rn(n0), __float2half_rn(n1));
    }
}

fp_rms_norm_kernel pick_rms_norm_kernel(const int blocks_per_warp)
{
    if (blocks_per_warp == 1) return rms_norm_kernel<1>;
    if (blocks_per_warp == 2) return rms_norm_kernel<2>;
    if (blocks_per_warp == 3) return rms_norm_kernel<3>;
    if (blocks_per_warp == 4) return rms_norm_kernel<4>;
    if (blocks_per_warp == 5) return rms_norm_kernel<5>;
    if (blocks_per_warp == 6) return rms_norm_kernel<6>;
    if (blocks_per_warp == 7) return rms_norm_kernel<7>;
    if (blocks_per_warp == 8) return rms_norm_kernel<8>;
	return NULL;
}

void rms_norm_cuda
(
    const half* x,
    const half* w,
    half* y,
    const float epsilon,
    const int rows,
    const int dim
)
{
    dim3 blockDim, gridDim;
    blockDim.x = NUM_THREADS;
    blockDim.y = 1;
    gridDim.x = rows;
    gridDim.y = 1;

    float r_dim = 1.0f / (float) dim;

    int blocks_per_warp = DIVIDE(dim, NUM_THREADS * 2);
    fp_rms_norm_kernel kernel = pick_rms_norm_kernel(blocks_per_warp);
    kernel<<<gridDim, blockDim>>>(x, w, y, epsilon, r_dim, rows, dim);
}
