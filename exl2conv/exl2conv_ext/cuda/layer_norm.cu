#include "hip/hip_runtime.h"
#include "layer_norm.cuh"
#include "util.cuh"
#include "compat.cuh"

#if defined(USE_ROCM)
#define NUM_WARPS (1024 / warpSize)
#define WARP_SIZE (warpSize)
#else
#define NUM_WARPS 32
#define WARP_SIZE 32
#endif

// y = x * w / sqrt(row_mean(x * x) + epsilon)

#define BLOCK_SIZE WARP_SIZE
#define NUM_THREADS (NUM_WARPS * WARP_SIZE)

typedef void (*fp_layer_norm_kernel)
(
    const half*,
    const half*,
    const half*,
    half*,
    const float,
    const float,
    const int,
    const int
);

template <int blocks_per_warp>
__global__ void layer_norm_kernel
(
    const half* __restrict__ x,
    const half* __restrict__ w,
    const half* __restrict__ b,
    half* __restrict__ y,
    const float epsilon,
    const float r_dim,
    const int rows,
    const int dim
)
{
    int warp_id = threadIdx.x / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int row = blockIdx.x;
    const half2* x_row = (const half2*) (x + row * dim);
    half2* y_row = (half2*) (y + row * dim);
    const half2* w2 = (const half2*) w;
    const half2* b2 = (const half2*) b;

    float itemf[blocks_per_warp][2];

    // Compute sum for each block

    float sum = 0.0f;

    #pragma unroll
    for (int i = 0; i < blocks_per_warp; i++)
    {
        int column = warp_id * WARP_SIZE + lane_id + NUM_THREADS * i;
        if (column >= dim / 2) break;

        half2 x2 = x_row[column];
        float f0 = __half2float(__low2half(x2));
        float f1 = __half2float(__high2half(x2));
        f0 = fmaxf(-65504.0f, fminf(f0, 65504.0f));
        f1 = fmaxf(-65504.0f, fminf(f1, 65504.0f));
        itemf[i][0] = f0;
        itemf[i][1] = f1;
        sum += f0;
        sum += f1;
    }

    // Shuffle to sum across lanes

    __shared__ float sums[NUM_WARPS];

    for(int offset = warpSize / 2; offset > 0; offset /= 2) sum += __shfl_xor_sync(0xffffffff, sum, offset);
    if (lane_id == 0) sums[warp_id] = sum;
    __syncthreads();

    // Load partial sums from across warps, shuffle again across lanes

    sum = sums[lane_id];
    for(int offset = warpSize / 2; offset > 0; offset /= 2) sum += __shfl_xor_sync(0xffffffff, sum, offset);

    // Compute mean

    float mean = sum * r_dim;

    // Compute square of distance to mean

    sum = 0.0f;

    #pragma unroll
    for (int i = 0; i < blocks_per_warp; i++)
    {
        int column = warp_id * WARP_SIZE + lane_id + NUM_THREADS * i;
        if (column >= dim / 2) break;

        float f0 = itemf[i][0];
        float f1 = itemf[i][1];
        f0 -= mean;
        f1 -= mean;
        itemf[i][0] = f0;
        itemf[i][1] = f1;
        sum = fma(f0, f0, sum);
        sum = fma(f1, f1, sum);
    }

    // Shuffle to sum across lanes

    for(int offset = warpSize / 2; offset > 0; offset /= 2) sum += __shfl_xor_sync(0xffffffff, sum, offset);
    if (lane_id == 0) sums[warp_id] = sum;
    __syncthreads();

    // Load partial sums from across warps, shuffle again across lanes

    sum = sums[lane_id];
    for(int offset = warpSize / 2; offset > 0; offset /= 2) sum += __shfl_xor_sync(0xffffffff, sum, offset);

    // Get 1/sqrt(variance)

    float rsvar = rsqrtf(sum * r_dim + epsilon);

    // Normalize x, scaling by w

    #pragma unroll 4
    for (int i = 0; i < blocks_per_warp; i++)
    {
        int column = warp_id * WARP_SIZE + lane_id + NUM_THREADS * i;
        if (column >= dim / 2) return;
        half2 w2_ = w2[column];

        float x_itemf0 = itemf[i][0];
        float x_itemf1 = itemf[i][1];
        float w_itemf0 = __half2float(__low2half(w2_));
        float w_itemf1 = __half2float(__high2half(w2_));
        float n0 = x_itemf0 * w_itemf0 * rsvar;
        float n1 = x_itemf1 * w_itemf1 * rsvar;
        half2 nh = __halves2half2(__float2half_rn(n0), __float2half_rn(n1));
        if (b) nh = __hadd2(nh, b2[column]);  // Optional bias

        y_row[column] = nh;
    }
}

fp_layer_norm_kernel pick_layer_norm_kernel(const int blocks_per_warp)
{
    if (blocks_per_warp == 1) return layer_norm_kernel<1>;
    if (blocks_per_warp == 2) return layer_norm_kernel<2>;
    if (blocks_per_warp == 3) return layer_norm_kernel<3>;
    if (blocks_per_warp == 4) return layer_norm_kernel<4>;
    if (blocks_per_warp == 5) return layer_norm_kernel<5>;
    if (blocks_per_warp == 6) return layer_norm_kernel<6>;
    if (blocks_per_warp == 7) return layer_norm_kernel<7>;
    if (blocks_per_warp == 8) return layer_norm_kernel<8>;
	return NULL;
}

void layer_norm_cuda
(
    const half* x,
    const half* w,
    const half* b,
    half* y,
    const float epsilon,
    const int rows,
    const int dim
)
{
    dim3 blockDim, gridDim;
    blockDim.x = NUM_THREADS;
    blockDim.y = 1;
    gridDim.x = rows;
    gridDim.y = 1;

    float r_dim = 1.0f / (float) dim;

    int blocks_per_warp = DIVIDE(dim, NUM_THREADS);
    fp_layer_norm_kernel kernel = pick_layer_norm_kernel(blocks_per_warp);
    kernel<<<gridDim, blockDim>>>(x, w, b, y, epsilon, r_dim, rows, dim);
}
